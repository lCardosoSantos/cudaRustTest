#include "hip/hip_runtime.h"
#include "testFields.cuh"

/**
 *  @brief Test self consistency in multiplication by constant:
 * 
 * 2(4x) = =8x
 * 2(2(2(2(2(2x))))) == 4(4(4x)) == 8(8x)
 * 3(4x) == 12(x)
 * 3(3(3(2(4(8x))))) == 12(12(12x))
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
template<typename T>
 __global__ void TestFieldMulConst(bool result, T *testval, const size_t testsize){
    //#warning Function not implemented: TestFieldMulConst
}

/**
 * @brief Multiplication test, using different values for different threads.
 * 
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
template<typename T>
 __global__ void TestFieldMul(bool result, T *testval, const size_t testsize){
    //#warning Function not implemented: TestFieldMul
}

/**
 * @brief Test for the commutative property of addition
 * 
 * x*y == y*x
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
template<typename T>
 __global__ void TestFieldCommutativeMul(bool result, T *testval, const size_t testsize){
    //#warning Function not implemented: TestFieldCommutativeMul
}

/**
 * @brief Test for the associative property of multiplication
 * 
 * (x*y)*z == x*(y*z)
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
template<typename T>
 __global__ void TestFieldAssociativeMul(bool result, T *testval, const size_t testsize){
    //#warning Function not implemented: TestFieldAssociativeMul
}
