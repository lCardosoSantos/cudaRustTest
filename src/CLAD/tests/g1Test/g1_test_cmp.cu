#include "hip/hip_runtime.h"
#include "g1_test.cuh"

/**
 * @brief Test for point copy
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool G1TestCpy(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: G1TestCpy
}

/**
 * @brief Test for point validation
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool G1TestIsPoint(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: G1TestIsPoint
}

/**
 * @brief Test for point equality
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool G1TestEq(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: G1TestEq
}

/**
 * @brief Test for point inequality
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool G1TestNeq(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: G1TestNeq
}

/**
 * @brief Test for point at infinity
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool G1TestIsInf(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: G1TestIsInf
}

