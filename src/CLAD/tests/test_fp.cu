#include "testUtil.cuh"
#include "testFields.cuh"
#include "fp.cuh"

#define TESTSIZE (size_t)256

__managed__  fp_t *testval_fp;

extern "C" void run_fp_tests(){\
    printf("\nFp tests\n");

    pass=false;
    hipError_t err;
    init(TESTSIZE, testval_fp);

    //commented tests are not implemented funcs

    //Linear time tests
    TEST_RUN(TestFieldCmp, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldMulConst, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldAdd, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldSub, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldMul, pass, testval_fp, TESTSIZE);

        // TEST_RUN(TestFieldSqr, pass, testval_fp, TESTSIZE);
        // TEST_RUN(TestFieldInv, pass, testval_fp, TESTSIZE);
        // TEST_RUN(TestFieldMMA, pass, testval_fp, TESTSIZE); 


    //Quadratic time tests
        // TEST_RUN(TestFieldSqr2, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldCommutativeAdd, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldCommutativeMul, pass, testval_fp, TESTSIZE);

    //Cubic time tests
    TEST_RUN(TestFieldAssociativeAdd, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldAssociativeMul, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldAddDistributiveLeft, pass, testval_fp, TESTSIZE);
    TEST_RUN(TestFieldAddDistributiveRight, pass, testval_fp, TESTSIZE);
        // TEST_RUN(TestFieldSubDistributiveLeft, pass, testval_fp, TESTSIZE);
        // TEST_RUN(TestFieldSubDistributiveRight, pass, testval_fp, TESTSIZE);

    printf("\n---\n");
    hipFree(testval_fp);
}

//Defined if the file is compiled for the Rust Library.
#ifndef RUST_TEST 

int main(int argc, char **argv){
    printf("Fp tests\n");
    run_fp_tests();
}

#endif
