#include "hip/hip_runtime.h"
#include "fp_test.cuh"

/**
 *  @brief Test self consistency in multiplication by constant:
 * 
 * 2(4x) = =8x
 * 2(2(2(2(2(2x))))) == 4(4(4x)) == 8(8x)
 * 3(4x) == 12(x)
 * 3(3(3(2(4(8x))))) == 12(12(12x))
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool FpTestMulConst(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: FpTestMulConst
}

/**
 * @brief Multiplication test, using different values for different threads.
 * 
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool FpTestMul(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: FpTestMul
}

/**
 * @brief Test for the commutative property of addition
 * 
 * x*y == y*x
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool FpTestCommutativeMul(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: FpTestCommutativeMul
}

/**
 * @brief Test for the associative property of multiplication
 * 
 * (x*y)*z == x*(y*z)
 * 
 * @param testval 
 * @param testsize 
 * 
 * @return bool 
 */
extern "C"__global__ bool FpTestAssociativeMul(bool result, testval_t *testval, const size_t testsize){
    #warning Function not implemented: FpTestAssociativeMul
}
