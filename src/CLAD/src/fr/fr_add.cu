#include "hip/hip_runtime.h"
// Copyright 2024 Dag Arne Osvik
// Copyright 2024 Luan Cardoso dos Santos

#include "fr.cuh"
#include "add320.cuh"
#include "fr_reduce5.cuh"

__noinline__ __device__ void fr_add(fr_t &z, const fr_t &x, const fr_t &y) {
    uint64_t
        z0, z1, z2, z3, z4,
        x0 = x[0], x1 = x[1], x2 = x[2], x3 = x[3],
        y0 = y[0], y1 = y[1], y2 = y[2], y3 = y[3];

    add320(z0, z1, z2, z3, z4,  x0, x1, x2, x3,  0,  y0, y1, y2, y3,  0);

    fr_reduce5(z0, z1, z2, z3, z0, z1, z2, z3, z4);

    z[0] = z0;
    z[1] = z1;
    z[2] = z2;
    z[3] = z3;
}

// vim: ts=4 et sw=4 si
